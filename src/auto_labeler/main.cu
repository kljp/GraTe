#include "../common/graph.h"
#include "bfs_al.cuh"
#include <sstream>
#include <fstream>
#include <climits>

template <typename vertex_t, typename index_t, typename depth_t>
void process_graph(

        std::string file_beg_pos,
        std::string file_adj_list,
        vertex_t INFTY
)
{

    const vertex_t gpu_id = 0;
    graph<vertex_t, index_t, double, vertex_t, index_t, double> *ginst
            = new graph<vertex_t, index_t, double, vertex_t, index_t, double>(file_beg_pos.c_str(), file_adj_list.c_str(), NULL);
    srand((unsigned int) wtime());

    vertex_t *src_list = new vertex_t[NUM_ITER];
    vertex_t src;
    for(int i = 0; i < NUM_ITER; i++){

        src = rand() % ginst->vert_count;

        if(ginst->beg_pos[src + 1] - ginst->beg_pos[src] > 0)
            src_list[i] = src;
        else
            i--;
    }

    bfs<vertex_t, index_t, depth_t>(

            src_list,
            ginst->beg_pos,
            ginst->csr,
            ginst->vert_count,
            ginst->edge_count,
            gpu_id,
            INFTY
    );

    delete[] src_list;
    delete ginst;
}

int main(int argc, char **argv){

    if(argc < 4){

        std::cout
                << "Required argument:\n"
                << "\t--csr : beg_pos and adj_list of input graph (e.g., --csr com-Orkut.mtx_beg_pos.bin com-Orkut.mtx_adj_list.bin)\n"
                << "Optional argument:\n"
                << "\t--verylarge : set data type of vertices and edges to ' unsigned long long' to handle very large input graph (e.g., com-Friendster), default='unsigned int'\n"
                << "\t--verbose : print breakdown of frontier processing techniques\n"
                << std::endl;

        exit(-1);
    }

    std::string file_beg_pos;
    std::string file_adj_list;
    bool is_verylarge = false;
    bool is_verbose = false;
    bool is_checked_input = false;
    bool is_checked_verylarge = false;
    bool is_checked_verbose = false;

    for(int i = 1; i < argc; i++){
        if(!strcmp(argv[i], "--csr") && i != argc - 1 && i != argc - 2){
            if(!is_checked_input){
                file_beg_pos = std::string(argv[i + 1]);
                file_adj_list = std::string(argv[i + 2]);
                is_checked_input = true;
            }
        }
        else if(!strcmp(argv[i], "--verylarge")){
            if(!is_checked_verylarge){
                is_verylarge = true;
                is_checked_verylarge = true;
            }
        }
        else if(!strcmp(argv[i], "--verbose")){
            if(!is_checked_verbose){
                is_verbose = true;
                is_checked_verbose = true;
            }
        }
    }

    if(is_verbose)
        verbose = true;
    else
        verbose = false;

    if(is_verylarge){
        std::cout << "Data type='unsigned long long'" << std::endl;
        process_graph<unsigned long long, unsigned long long, unsigned int>(
                file_beg_pos,
                file_adj_list,
                ULLONG_MAX
        );
    }
    else{
        std::cout << "Data type='unsigned int'" << std::endl;
        process_graph<unsigned int, unsigned int, unsigned int>(
                file_beg_pos,
                file_adj_list,
                UINT_MAX
        );
    }

    return 0;
}
